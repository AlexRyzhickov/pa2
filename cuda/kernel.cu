#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <stdio.h>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__device__ __host__ long int calc(long int n, long int m) {
    if (m > n) {
        return 0;
    }
    if (m == 1) {
        return 1;
    }
    if (n == m) {
        return 1;
    }
    if (m == 2) {
        return n / 2;
    }
    if (m == 3) {
        return (n * n + 3) / 12;
    }
    return calc(n - m, m) + calc(n - 1, m - 1);
}

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = calc(a[i], b[i]);
}

int main()
{
    const int n = 100;
    const int m = 17;
    const int arraySize = m;
    int a[arraySize] = { };
    int b[arraySize] = { };
    int c[arraySize] = { };

    for (int i = arraySize; i > 0; i--)
    {
        a[i - 1] = n - m;
        b[i - 1] = i;
    }

    time_t start, end;

    time(&start);
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    time(&end);

    int count = 0;
    for (int i = 0; i < arraySize; i++) {
        count = count + c[i];
    }

    printf("Count: %d \n", count);
    printf("The time: %f seconds\n", difftime(end, start));

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
